// Sample from NVIDIA:
// https://github.com/NVIDIA/cuda-samples/blob/master/Samples/0_Introduction/vectorAdd/vectorAdd.cu

#include <stdexcept>
#include <stdio.h>
#include <string>
#include <unistd.h>

// For the CUDA runtime routines (prefixed with "cuda_")
#include <hip/hip_runtime.h>

// Code to be executed in the GPU. Allows managing the number of loops to have
// an increased execution time.
__global__ void vectorSumKernel(const float *A, const float *B, float *C,
								int numElements, int loops) {
	int i = blockDim.x * blockIdx.x + threadIdx.x;

	for (size_t loopIdx = 0; loopIdx < loops; loopIdx++) {
		if (i < numElements) {
			C[i] = A[i] + B[i] + 0.0f;
		}
	}
}

int main(int argc, const char **argv) {
	// Error code to check return values for CUDA calls
	hipError_t err = hipSuccess;

	if (argc != 4) {
		fprintf(stderr, "Usage: %s <numElements> <loops> <waitTimeSeconds>\n",
				argv[0]);
		exit(EXIT_FAILURE);
	}

	int numElements, loops, waitTimeSeconds;

	try {
		numElements = std::stoi(argv[1]);
		loops = std::stoi(argv[2]);
		waitTimeSeconds = std::stoi(argv[3]);
	} catch (const std::invalid_argument &e) {
		fprintf(stderr, "Invalid argument: %s\n", e.what());
		exit(EXIT_FAILURE);
	}

	printf("Will wait %d seconds before starting...\n", waitTimeSeconds);
	sleep(waitTimeSeconds);

	// Print the vector length to be used, and compute its size
	size_t size = numElements * sizeof(float);
	printf("Vector size: %d elements (%zu bytes)\n", numElements, size);

	float *h_A = (float *)malloc(size);
	float *h_B = (float *)malloc(size);
	float *h_C = (float *)malloc(size);

	if (h_A == NULL || h_B == NULL || h_C == NULL) {
		fprintf(stderr, "Failed to allocate host vectors!\n");
		exit(EXIT_FAILURE);
	}

	// Initialize the host input vectors
	for (int i = 0; i < numElements; ++i) {
		h_A[i] = rand() / (float)RAND_MAX;
		h_B[i] = rand() / (float)RAND_MAX;
	}

	// Allocate the device input vector A
	float *d_A = NULL;
	err = hipMalloc((void **)&d_A, size);

	if (err != hipSuccess) {
		fprintf(stderr, "Failed to allocate device vector A (error code %s)!\n",
				hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}

	// Allocate the device input vector B
	float *d_B = NULL;
	err = hipMalloc((void **)&d_B, size);

	if (err != hipSuccess) {
		fprintf(stderr, "Failed to allocate device vector B (error code %s)!\n",
				hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}

	// Allocate the device output vector C
	float *d_C = NULL;
	err = hipMalloc((void **)&d_C, size);

	if (err != hipSuccess) {
		fprintf(stderr, "Failed to allocate device vector C (error code %s)!\n",
				hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}

	// Copy the host input vectors A and B in host memory to the device input
	// vectors in
	// device memory
	printf("Copy input data from the host memory to the CUDA device\n");
	err = hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);

	if (err != hipSuccess) {
		fprintf(
			stderr,
			"Failed to copy vector A from host to device (error code %s)!\n",
			hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}

	err = hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

	if (err != hipSuccess) {
		fprintf(
			stderr,
			"Failed to copy vector B from host to device (error code %s)!\n",
			hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}

	// Launch the Vector Add CUDA Kernel
	int threadsPerBlock = 256;
	int blocksPerGrid = (numElements + threadsPerBlock - 1) / threadsPerBlock;
	printf("CUDA kernel launch with %d blocks of %d threads\n", blocksPerGrid,
		   threadsPerBlock);
	vectorSumKernel<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C,
														numElements, loops);
	err = hipGetLastError();

	if (err != hipSuccess) {
		fprintf(stderr,
				"Failed to launch vectorSumKernel kernel (error code %s)!\n",
				hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}

	// Copy the device result vector in device memory to the host result vector
	// in host memory.
	printf("Copy output data from the CUDA device to the host memory\n");
	err = hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);

	if (err != hipSuccess) {
		fprintf(
			stderr,
			"Failed to copy vector C from device to host (error code %s)!\n",
			hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}

	// Verify that the result vector is correct
	for (int i = 0; i < numElements; ++i) {
		if (fabs(h_A[i] + h_B[i] - h_C[i]) > 1e-5) {
			fprintf(stderr, "Result verification failed at element %d!\n", i);
			exit(EXIT_FAILURE);
		}
	}

	printf("Test PASSED\n");

	// Free device global memory
	err = hipFree(d_A);

	if (err != hipSuccess) {
		fprintf(stderr, "Failed to free device vector A (error code %s)!\n",
				hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}

	err = hipFree(d_B);

	if (err != hipSuccess) {
		fprintf(stderr, "Failed to free device vector B (error code %s)!\n",
				hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}

	err = hipFree(d_C);

	if (err != hipSuccess) {
		fprintf(stderr, "Failed to free device vector C (error code %s)!\n",
				hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}

	// Free host memory
	free(h_A);
	free(h_B);
	free(h_C);

	printf("Done\n");
	return 0;
}
